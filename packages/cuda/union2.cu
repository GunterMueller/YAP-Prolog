#include <thrust/device_vector.h>
#include <thrust/unique.h>
#include <thrust/distance.h>
#include <iostream>

typedef struct n2
{
	int v[2];
}s2;

typedef struct n3
{
	int v[3];
}s3;

struct p2
{
	__host__ __device__
    	bool operator()(const s2 &r1, const s2 &r2)
    	{
     		int x;
		for(x = 0; x < 2; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o2
{
	__host__ __device__
    	bool operator()(const s2 &r1, const s2 &r2)
    	{
     		int x;
		for(x = 0; x < 2; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

struct p3
{
	__host__ __device__
    	bool operator()(const s3 &r1, const s3 &r2)
    	{
     		int x;
		for(x = 0; x < 3; x++)
		{
			if(r1.v[x] != r2.v[x])
				return false;
		}
		return true;
    	}
};

struct o3
{
	__host__ __device__
    	bool operator()(const s3 &r1, const s3 &r2)
    	{
     		int x;
		for(x = 0; x < 3; x++)
		{
			if(r1.v[x] > r2.v[x])
				return true;
			if(r1.v[x] < r2.v[x])
				return false;
		}
		return false;
    	}
};

int unir(int *res, int rows, int tipo)
{
	thrust::device_ptr<int> pt, re;
	thrust::device_ptr<s2> pt2, re2;
	thrust::device_ptr<s3> pt3, re3;
	s2 *t2;
	s3 *t3;
	int flag, nrows;

#if TIMER
	cuda_stats.unions++;
#endif
	switch(tipo)
	{
		case 1: 
		{
			pt = thrust::device_pointer_cast(res);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt, pt + rows);
					re = thrust::unique(pt, pt + rows);
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir");
				}				
			}
			nrows = thrust::distance(pt, re);
			thrust::device_vector<int> iVec(pt, pt + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;	
		}			
		case 2: 
		{
			t2 = (s2*)res;
			
			/*int *a, x, y;
			a = (int *)malloc(rows * 2 * sizeof(int));
			hipMemcpy(a, res, rows * 2 * sizeof(int), hipMemcpyDeviceToHost);
			cout << "INI" << endl;
			for(x = 0; x < rows; x++)
			{
				for(y = 0; y < 2; y++)
					cout << a[x * 2 + y] << " ";
				cout << endl;	
			}
			cout << "INI fin" << endl;
			free(a);*/

			pt2 = thrust::device_pointer_cast(t2);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt2, pt2 + rows, o2());
					re2 = thrust::unique(pt2, pt2 + rows, p2());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir");
				}				
			}
			nrows = thrust::distance(pt2, re2);
			thrust::device_vector<s2> iVec(pt2, pt2 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();

			/*tam = (int)(re2.get() - pt2.get());
			a = (int *)malloc(tam * 2 * sizeof(int));
			hipMemcpy(a, res, tam * 2 * sizeof(int), hipMemcpyDeviceToHost);
			cout << "FIN" << endl;
			for(x = 0; x < tam; x++)
			{
				for(y = 0; y < 2; y++)
					cout << a[x * 2 + y] << " ";
				cout << endl;	
			}
			cout << "FIN fin" << endl;
			free(a);
			cout << "antes = " << rows << " despues = " << thrust::distance(pt2, re2) << endl;*/

			return nrows;
		}
		case 3: 
		{
			t3 = (s3*)res;
			pt3 = thrust::device_pointer_cast(t3);
			flag = 0;
			while(flag != 1)
			{
				try
				{
					thrust::sort(pt3, pt3 + rows, o3());
					re3 = thrust::unique(pt3, pt3 + rows, p3());
					flag = 1;
				}
				catch(std::bad_alloc &e)
				{
					limpiar("sort/unique in unir");
				}				
			}
			nrows = thrust::distance(pt3, re3);
			thrust::device_vector<s3> iVec(pt3, pt3 + rows);
			iVec.resize(nrows);
			iVec.shrink_to_fit();
			return nrows;
		}
	}
	return 0;
}
